
#include <hip/hip_runtime.h>

__device__
void clamp_pars_4_f(const float* lower_bound, const float* upper_bound, float* pars, int tid, int N) 
{
	for (int i = 0; i < 4; ++i) {
		int index = i*N+tid;
		float p = pars[index];
		float u = upper_bound[index];
		float l = lower_bound[index];

		if (p > u) {
			pars[index] = u;
		} else if (p < l) {
			pars[index] = l;
		}
	}
}

extern "C" __global__
void k_clamp_pars_4_f(const float* lower_bound, const float* upper_bound, float* pars, int N) 
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < N) {
		clamp_pars_4_f(lower_bound, upper_bound, pars, tid, N);
	}
}
