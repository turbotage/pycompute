
#include <hip/hip_runtime.h>

extern "C" __global__
void eval_jac_hes_21000000_4_1_f_f7def86f0a03(const float* params, const float* consts, float* eval, 
	float* jac, float* hes, unsigned int N) 
{
	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < N) {

		float pars[4];
		for (int i = 0; i < 4; ++i) {
			pars[i] = params[i*21000000+tid];
		}

		float x0 = expf(-pars[2]*consts[0*21000000+tid]);
		float x1 = x0*pars[1];
		float x2 = expf(-pars[3]*consts[0*21000000+tid]);
		float x3 = x2*(1 - pars[1]);
		float x4 = x1 + x3;
		float x5 = x0 - x2;
		float x6 = pars[0]*consts[0*21000000+tid];
		float x7 = x3*consts[0*21000000+tid];
		float x8 = pars[0]*powf(consts[0*21000000+tid], 2);


		eval[tid] = x4*pars[0];

		jac[0*21000000+tid] = x4;
		jac[1*21000000+tid] = x5*pars[0];
		jac[2*21000000+tid] = -x1*x6;
		jac[3*21000000+tid] = -x7*pars[0];


		hes[1*21000000+tid] = x5;
		hes[3*21000000+tid] = -x1*consts[0*21000000+tid];
		hes[4*21000000+tid] = -x0*x6;
		hes[5*21000000+tid] = x1*x8;
		hes[6*21000000+tid] = -x7;
		hes[7*21000000+tid] = x2*x6;
		hes[9*21000000+tid] = x3*x8;


	}


}
